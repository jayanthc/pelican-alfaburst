#include "hip/hip_runtime.h"
#ifndef DEDISPERSE_KERNEL_H_
#define DEDISPERSE_KERNEL_H_
#include <assert.h>
#include <iostream>
#include "stdio.h"
#include "DedispersionParameters.h"

// Stores temporary shift values
//__device__ __constant__ float dm_shifts[8192];
//__device__ __constant__ int   i_nsamp, i_maxshift, i_nchans;
//__device__ __shared__ float f_line[ARRAYSIZE];


//{{{ global_for_time_dedisperse_loop
__global__ void cache_dedisperse_loop(float *outbuff, float *buff, float mstartdm,
                                      float mdmstep, const float* dm_shifts,
                                      const int i_nsamp, const int i_maxshift,
                                      const int i_nchans )
{

    int   shift;
    float local_kernel_t[NUMREG];

    int t  = blockIdx.x * NUMREG * DIVINT  + threadIdx.x;

    // Initialise the time accumulators
    for(int i = 0; i < NUMREG; i++) local_kernel_t[i] = 0.0f;

    float shift_temp = mstartdm + ((blockIdx.y * DIVINDM + threadIdx.y) * mdmstep);

    // Loop over the frequency channels.
    for(int c = 0; c < i_nchans; c++) {
        // Calculate the initial shift for this given frequency
        // channel (c) at the current despersion measure (dm) 
        // ** dm is constant for this thread!!**
        shift = (c * i_nsamp + t) + __float2int_rz (dm_shifts[c] * shift_temp);

        #pragma unroll
        for(int i = 0; i < NUMREG; i++) {
            local_kernel_t[i] += buff[shift + (i * DIVINT) ];
            //local_kernel_t[i] += __ldg(&buff[shift + (i * DIVINT) ]);
        }
    }

    // Write the accumulators to the output array. 
    #pragma unroll
    for(int i = 0; i < NUMREG; i++) {
        outbuff[((blockIdx.y * DIVINDM) + threadIdx.y)* (i_nsamp-i_maxshift) + (i * DIVINT) + (NUMREG * DIVINT * blockIdx.x) + threadIdx.x] = local_kernel_t[i];
    }
}

/// C Wrapper for brute-force algo
extern "C" void cacheDedisperseLoop( float *outbuff, long outbufSize, float *buff, float mstartdm,
                                     float mdmstep, int tdms, int numSamples,
                                     const float* dmShift,
                                     const int maxshift,
                                     const int i_nchans ) {

    hipMemset(outbuff, 0, outbufSize );
    int divisions_in_t  = DIVINT;
    int divisions_in_dm = DIVINDM - (tdms%DIVINDM); // ensure divides exactly into
                                                    // our dm parameter space
    int num_reg = NUMREG;
    int num_blocks_t = (numSamples - maxshift)/(divisions_in_t * num_reg);
    int num_blocks_dm = tdms/divisions_in_dm;

/*
    std::cout << "\nnumSamples\t" << numSamples << std::endl;
    std::cout << "\ndivisions_in_t\t" << divisions_in_t << std::endl;
    std::cout << "\ndivisions_in_dm\t" << divisions_in_dm << std::endl;
    std::cout << "\nnum_reg\t" << num_reg << std::endl;
    std::cout << "\nnum_blocks_t\t" << num_blocks_t << std::endl;
    std::cout << "\nnum_blocks_dm\t" << num_blocks_dm << std::endl;
    std::cout << "\ntdms\t" << tdms << std::endl;
    std::cout << "mdmstep\t" << mdmstep << std::endl;
    std::cout << "mstartdm\t" << mstartdm << std::endl;
    std::cout << "buff\t" << buff << std::endl;
    std::cout << "outbuff\t" << outbuff << std::endl;
*/

    dim3 threads_per_block(divisions_in_t, divisions_in_dm);
    dim3 num_blocks(num_blocks_t,num_blocks_dm);

    cache_dedisperse_loop<<< num_blocks, threads_per_block >>>( outbuff, buff, 
                mstartdm, mdmstep, dmShift, numSamples, maxshift, i_nchans );
}

#endif
